#include "hip/hip_runtime.h"
//===--- omptarget-nvptx.cu - NVPTX OpenMP GPU initialization ---- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the initialization code for the GPU
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// global data tables
////////////////////////////////////////////////////////////////////////////////

extern __device__
    omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext, OMP_STATE_COUNT>
        omptarget_nvptx_device_State[MAX_SM];

extern __device__ __shared__
    omptarget_nvptx_ThreadPrivateContext *omptarget_nvptx_threadPrivateContext;

//
// The team master sets the outlined function and its arguments in these
// variables to communicate with the workers.  Since they are in shared memory,
// there is one copy of these variables for each kernel, instance, and team.
//
extern volatile __device__ __shared__ omptarget_nvptx_WorkFn
    omptarget_nvptx_workFn;
extern __device__ __shared__ uint32_t execution_param;

////////////////////////////////////////////////////////////////////////////////
// init entry points
////////////////////////////////////////////////////////////////////////////////

INLINE unsigned smid() {
  unsigned id;
  asm("mov.u32 %0, %%smid;" : "=r"(id));
  return id;
}

EXTERN void __kmpc_kernel_init_params(void *Ptr) {
  PRINT(LD_IO, "call to __kmpc_kernel_init_params with version %f\n",
        OMPTARGET_NVPTX_VERSION);

  SetTeamsReductionScratchpadPtr(Ptr);
}

EXTERN void __kmpc_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime) {
  PRINT(LD_IO, "call to __kmpc_kernel_init with version %f\n",
        OMPTARGET_NVPTX_VERSION);

  if (!RequiresOMPRuntime) {
    // If OMP runtime is not required don't initialize OMP state.
    setExecutionParameters(Generic, RuntimeUninitialized);
    return;
  }
  setExecutionParameters(Generic, RuntimeInitialized);

  int threadIdInBlock = GetThreadIdInBlock();
  ASSERT0(LT_FUSSY, threadIdInBlock == GetMasterThreadID(),
          "__kmpc_kernel_init() must be called by team master warp only!");
  PRINT0(LD_IO, "call to __kmpc_kernel_init for master\n");

  // Get a state object from the queue.
  int slot = smid() % MAX_SM;
  omptarget_nvptx_threadPrivateContext =
      omptarget_nvptx_device_State[slot].Dequeue();
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  omptarget_nvptx_threadPrivateContext->SetSourceQueue(slot);
#endif

  // init thread private
  int threadId = GetLogicalThreadIdInBlock();
  omptarget_nvptx_threadPrivateContext->InitThreadPrivateContext(threadId);

  // init team context
  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  currTeamDescr.InitTeamDescr();
  // this thread will start execution... has to update its task ICV
  // to point to the level zero task ICV. That ICV was init in
  // InitTeamDescr()
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(
      threadId, currTeamDescr.LevelZeroTaskDescr());

  // set number of threads and thread limit in team to started value
  omptarget_nvptx_TaskDescr *currTaskDescr =
      omptarget_nvptx_threadPrivateContext->GetTopLevelTaskDescr(threadId);
  currTaskDescr->NThreads() = GetNumberOfWorkersInTeam();
  currTaskDescr->ThreadLimit() = ThreadLimit;
}

EXTERN void __kmpc_kernel_deinit(int16_t IsOMPRuntimeInitialized) {
  if (IsOMPRuntimeInitialized) {
    // Enqueue omp state object for use by another team.
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
    int slot = omptarget_nvptx_threadPrivateContext->GetSourceQueue();
#else
    int slot = smid() % MAX_SM;
#endif
    omptarget_nvptx_device_State[slot].Enqueue(
        omptarget_nvptx_threadPrivateContext);
  }
  // Done with work.  Kill the workers.
  omptarget_nvptx_workFn = 0;
}

EXTERN void __kmpc_spmd_kernel_init(int ThreadLimit, int16_t RequiresOMPRuntime,
                                    int16_t RequiresDataSharing) {
  PRINT0(LD_IO, "call to __kmpc_spmd_kernel_init\n");

  if (!RequiresOMPRuntime) {
    // If OMP runtime is not required don't initialize OMP state.
    setExecutionParameters(Spmd, RuntimeUninitialized);
    return;
  }
  setExecutionParameters(Spmd, RuntimeInitialized);

  //
  // Team Context Initialization.
  //
  // In SPMD mode there is no master thread so use any cuda thread for team
  // context initialization.
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Get a state object from the queue.
    int slot = smid() % MAX_SM;
    omptarget_nvptx_threadPrivateContext =
        omptarget_nvptx_device_State[slot].Dequeue();

    omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
    omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();
    // init team context
    currTeamDescr.InitTeamDescr();
    // init counters (copy start to init)
    workDescr.CounterGroup().Reset();
  }
  __syncthreads();

  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();

  //
  // Initialize task descr for each thread.
  //
  omptarget_nvptx_TaskDescr *newTaskDescr =
      omptarget_nvptx_threadPrivateContext->Level1TaskDescr(threadId);
  ASSERT0(LT_FUSSY, newTaskDescr, "expected a task descr");
  newTaskDescr->InitLevelOneTaskDescr(ThreadLimit,
                                      currTeamDescr.LevelZeroTaskDescr());
  newTaskDescr->ThreadLimit() = ThreadLimit;
  // install new top descriptor
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(threadId,
                                                             newTaskDescr);

  // init thread private from init value
  workDescr.CounterGroup().Init(
      omptarget_nvptx_threadPrivateContext->Priv(threadId));
  PRINT(LD_PAR,
        "thread will execute parallel region with id %d in a team of "
        "%d threads\n",
        newTaskDescr->ThreadId(), newTaskDescr->ThreadsInTeam());

  if (RequiresDataSharing && threadId % WARPSIZE == 0) {
    // Warp master innitializes data sharing environment.
    unsigned WID = threadId / WARPSIZE;
    __kmpc_data_sharing_slot *RootS = currTeamDescr.RootS(
        WID, WID == WARPSIZE - 1);
    DataSharingState.SlotPtr[WID] = RootS;
    DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];
  }
}

EXTERN void __kmpc_spmd_kernel_deinit() {
  // We're not going to pop the task descr stack of each thread since
  // there are no more parallel regions in SPMD mode.
  __syncthreads();
  int threadId = GetThreadIdInBlock();
  if (threadId == 0) {
    // Enqueue omp state object for use by another team.
    int slot = smid() % MAX_SM;
    omptarget_nvptx_device_State[slot].Enqueue(
        omptarget_nvptx_threadPrivateContext);
  }
}

// Return true if the current target region is executed in SPMD mode.
EXTERN int8_t __kmpc_is_spmd_exec_mode() {
  return isSPMDMode();
}
