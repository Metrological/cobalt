#include "hip/hip_runtime.h"
//===---- reduction.cu - NVPTX OpenMP reduction implementation ---- CUDA
//-*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of reduction with KMPC interface.
//
//===----------------------------------------------------------------------===//

#include <complex.h>
#include <stdio.h>

#include "omptarget-nvptx.h"

// may eventually remove this
EXTERN
int32_t __gpu_block_reduce() {
  int tid = GetLogicalThreadIdInBlock();
  int nt = GetNumberOfOmpThreads(tid, isSPMDMode(), isRuntimeUninitialized());
  if (nt != blockDim.x)
    return 0;
  unsigned tnum = __ACTIVEMASK();
  if (tnum != (~0x0)) // assume swapSize is 32
    return 0;
  return 1;
}

EXTERN
int32_t __kmpc_reduce_gpu(kmp_Indent *loc, int32_t global_tid, int32_t num_vars,
                          size_t reduce_size, void *reduce_data,
                          void *reduce_array_size, kmp_ReductFctPtr *reductFct,
                          kmp_CriticalName *lck) {
  int threadId = GetLogicalThreadIdInBlock();
  omptarget_nvptx_TaskDescr *currTaskDescr = getMyTopTaskDescriptor(threadId);
  int numthread;
  if (currTaskDescr->IsParallelConstruct()) {
    numthread =
        GetNumberOfOmpThreads(threadId, isSPMDMode(), isRuntimeUninitialized());
  } else {
    numthread = GetNumberOfOmpTeams();
  }

  if (numthread == 1)
    return 1;
  if (!__gpu_block_reduce())
    return 2;
  if (threadIdx.x == 0)
    return 1;
  return 0;
}

EXTERN
int32_t __kmpc_reduce_combined(kmp_Indent *loc) {
  return threadIdx.x == 0 ? 2 : 0;
}

EXTERN
int32_t __kmpc_reduce_simd(kmp_Indent *loc) {
  return (threadIdx.x % 32 == 0) ? 1 : 0;
}

EXTERN
void __kmpc_nvptx_end_reduce(int32_t global_tid) {}

EXTERN
void __kmpc_nvptx_end_reduce_nowait(int32_t global_tid) {}

EXTERN int32_t __kmpc_shuffle_int32(int32_t val, int16_t delta, int16_t size) {
  return __SHFL_DOWN_SYNC(0xFFFFFFFF, val, delta, size);
}

EXTERN int64_t __kmpc_shuffle_int64(int64_t val, int16_t delta, int16_t size) {
  int lo, hi;
  asm volatile("mov.b64 {%0,%1}, %2;" : "=r"(lo), "=r"(hi) : "l"(val));
  hi = __SHFL_DOWN_SYNC(0xFFFFFFFF, hi, delta, size);
  lo = __SHFL_DOWN_SYNC(0xFFFFFFFF, lo, delta, size);
  asm volatile("mov.b64 %0, {%1,%2};" : "=l"(val) : "r"(lo), "r"(hi));
  return val;
}

static INLINE void gpu_regular_warp_reduce(void *reduce_data,
                                           kmp_ShuffleReductFctPtr shflFct) {
  for (uint32_t mask = WARPSIZE / 2; mask > 0; mask /= 2) {
    shflFct(reduce_data, /*LaneId - not used= */ 0,
            /*Offset = */ mask, /*AlgoVersion=*/0);
  }
}

static INLINE void gpu_irregular_warp_reduce(void *reduce_data,
                                             kmp_ShuffleReductFctPtr shflFct,
                                             uint32_t size, uint32_t tid) {
  uint32_t curr_size;
  uint32_t mask;
  curr_size = size;
  mask = curr_size / 2;
  while (mask > 0) {
    shflFct(reduce_data, /*LaneId = */ tid, /*Offset=*/mask, /*AlgoVersion=*/1);
    curr_size = (curr_size + 1) / 2;
    mask = curr_size / 2;
  }
}

static INLINE uint32_t
gpu_irregular_simd_reduce(void *reduce_data, kmp_ShuffleReductFctPtr shflFct) {
  uint32_t lanemask_lt;
  uint32_t lanemask_gt;
  uint32_t size, remote_id, physical_lane_id;
  physical_lane_id = GetThreadIdInBlock() % WARPSIZE;
  asm("mov.u32 %0, %%lanemask_lt;" : "=r"(lanemask_lt));
  uint32_t Liveness = __BALLOT_SYNC(0xFFFFFFFF, true);
  uint32_t logical_lane_id = __popc(Liveness & lanemask_lt) * 2;
  asm("mov.u32 %0, %%lanemask_gt;" : "=r"(lanemask_gt));
  do {
    Liveness = __BALLOT_SYNC(0xFFFFFFFF, true);
    remote_id = __ffs(Liveness & lanemask_gt);
    size = __popc(Liveness);
    logical_lane_id /= 2;
    shflFct(reduce_data, /*LaneId =*/logical_lane_id,
            /*Offset=*/remote_id - 1 - physical_lane_id, /*AlgoVersion=*/2);
  } while (logical_lane_id % 2 == 0 && size > 1);
  return (logical_lane_id == 0);
}

EXTERN
int32_t __kmpc_nvptx_simd_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                        size_t reduce_size, void *reduce_data,
                                        kmp_ShuffleReductFctPtr shflFct,
                                        kmp_InterWarpCopyFctPtr cpyFct) {
  uint32_t Liveness = __BALLOT_SYNC(0xFFFFFFFF, true);
  if (Liveness == 0xffffffff) {
    gpu_regular_warp_reduce(reduce_data, shflFct);
    return GetThreadIdInBlock() % WARPSIZE ==
           0; // Result on lane 0 of the simd warp.
  } else {
    return gpu_irregular_simd_reduce(
        reduce_data, shflFct); // Result on the first active lane.
  }
}

INLINE
int32_t nvptx_parallel_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                     size_t reduce_size, void *reduce_data,
                                     kmp_ShuffleReductFctPtr shflFct,
                                     kmp_InterWarpCopyFctPtr cpyFct,
                                     bool isSPMDExecutionMode,
                                     bool isRuntimeUninitialized = false) {
  uint32_t BlockThreadId = GetLogicalThreadIdInBlock();
  uint32_t NumThreads = GetNumberOfOmpThreads(
      BlockThreadId, isSPMDExecutionMode, isRuntimeUninitialized);
  if (NumThreads == 1)
    return 1;
  /*
   * This reduce function handles reduction within a team. It handles
   * parallel regions in both L1 and L2 parallelism levels. It also
   * supports Generic, SPMD, and NoOMP modes.
   *
   * 1. Reduce within a warp.
   * 2. Warp master copies value to warp 0 via shared memory.
   * 3. Warp 0 reduces to a single value.
   * 4. The reduced value is available in the thread that returns 1.
   */

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  uint32_t WarpsNeeded = (NumThreads + WARPSIZE - 1) / WARPSIZE;
  uint32_t WarpId = BlockThreadId / WARPSIZE;

  // Volta execution model:
  // For the Generic execution mode a parallel region either has 1 thread and
  // beyond that, always a multiple of 32. For the SPMD execution mode we may
  // have any number of threads.
  if ((NumThreads % WARPSIZE == 0) || (WarpId < WarpsNeeded - 1))
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (NumThreads > 1) // Only SPMD execution mode comes thru this case.
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/NumThreads % WARPSIZE,
                              /*LaneId=*/GetThreadIdInBlock() % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  //
  // Only L1 parallel region can enter this if condition.
  if (NumThreads > WARPSIZE) {
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                BlockThreadId);

    return BlockThreadId == 0;
  }
  return BlockThreadId == 0;
#else
  uint32_t Liveness = __BALLOT_SYNC(0xFFFFFFFF, true);
  if (Liveness == 0xffffffff) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (!(Liveness & (Liveness + 1))) // Partial warp but contiguous lanes
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/__popc(Liveness),
                              /*LaneId=*/GetThreadIdInBlock() % WARPSIZE);
  else if (!isRuntimeUninitialized) // Dispersed lanes. Only threads in L2
                                    // parallel region may enter here; return
                                    // early.
    return gpu_irregular_simd_reduce(reduce_data, shflFct);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  //
  // Only L1 parallel region can enter this if condition.
  if (NumThreads > WARPSIZE) {
    uint32_t WarpsNeeded = (NumThreads + WARPSIZE - 1) / WARPSIZE;
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    uint32_t WarpId = BlockThreadId / WARPSIZE;
    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded,
                                BlockThreadId);

    return BlockThreadId == 0;
  } else if (isRuntimeUninitialized /* Never an L2 parallel region without the OMP runtime */) {
    return BlockThreadId == 0;
  }

  // Get the OMP thread Id. This is different from BlockThreadId in the case of
  // an L2 parallel region.
  return GetOmpThreadId(BlockThreadId, isSPMDExecutionMode,
                        isRuntimeUninitialized) == 0;
#endif // __CUDA_ARCH__ >= 700
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(global_tid, num_vars, reduce_size,
                                      reduce_data, shflFct, cpyFct,
                                      /*isSPMDExecutionMode=*/isSPMDMode());
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_simple_spmd(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(global_tid, num_vars, reduce_size,
                                      reduce_data, shflFct, cpyFct,
                                      /*isSPMDExecutionMode=*/true,
                                      /*isRuntimeUninitialized=*/true);
}

EXTERN
int32_t __kmpc_nvptx_parallel_reduce_nowait_simple_generic(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct) {
  return nvptx_parallel_reduce_nowait(global_tid, num_vars, reduce_size,
                                      reduce_data, shflFct, cpyFct,
                                      /*isSPMDExecutionMode=*/false,
                                      /*isRuntimeUninitialized=*/true);
}

INLINE
int32_t nvptx_teams_reduce_nowait(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    kmp_CopyToScratchpadFctPtr scratchFct, kmp_LoadReduceFctPtr ldFct,
    bool isSPMDExecutionMode, bool isRuntimeUninitialized = false) {
  uint32_t ThreadId = GetLogicalThreadIdInBlock();
  // In non-generic mode all workers participate in the teams reduction.
  // In generic mode only the team master participates in the teams
  // reduction because the workers are waiting for parallel work.
  uint32_t NumThreads =
      isSPMDExecutionMode
          ? GetNumberOfOmpThreads(ThreadId, /*isSPMDExecutionMode=*/true,
                                  isRuntimeUninitialized)
          : /*Master thread only*/ 1;
  uint32_t TeamId = GetBlockIdInKernel();
  uint32_t NumTeams = GetNumberOfBlocksInKernel();
  __shared__ volatile bool IsLastTeam;

  // Team masters of all teams write to the scratchpad.
  if (ThreadId == 0) {
    unsigned int *timestamp = GetTeamsReductionTimestamp();
    char *scratchpad = GetTeamsReductionScratchpad();

    scratchFct(reduce_data, scratchpad, TeamId, NumTeams);
    __threadfence();

    // atomicInc increments 'timestamp' and has a range [0, NumTeams-1].
    // It resets 'timestamp' back to 0 once the last team increments
    // this counter.
    unsigned val = atomicInc(timestamp, NumTeams - 1);
    IsLastTeam = val == NumTeams - 1;
  }

  // We have to wait on L1 barrier because in GENERIC mode the workers
  // are waiting on barrier 0 for work.
  //
  // If we guard this barrier as follows it leads to deadlock, probably
  // because of a compiler bug: if (!IsGenericMode()) __syncthreads();
  uint16_t SyncWarps = (NumThreads + WARPSIZE - 1) / WARPSIZE;
  named_sync(L1_BARRIER, SyncWarps * WARPSIZE);

  // If this team is not the last, quit.
  if (/* Volatile read by all threads */ !IsLastTeam)
    return 0;

    //
    // Last team processing.
    //

    // Threads in excess of #teams do not participate in reduction of the
    // scratchpad values.
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  uint32_t ActiveThreads = NumThreads;
  if (NumTeams < NumThreads) {
    ActiveThreads =
        (NumTeams < WARPSIZE) ? 1 : NumTeams & ~((uint16_t)WARPSIZE - 1);
  }
  if (ThreadId >= ActiveThreads)
    return 0;

  // Load from scratchpad and reduce.
  char *scratchpad = GetTeamsReductionScratchpad();
  ldFct(reduce_data, scratchpad, ThreadId, NumTeams, /*Load only*/ 0);
  for (uint32_t i = ActiveThreads + ThreadId; i < NumTeams; i += ActiveThreads)
    ldFct(reduce_data, scratchpad, i, NumTeams, /*Load and reduce*/ 1);

  uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
  uint32_t WarpId = ThreadId / WARPSIZE;

  // Reduce across warps to the warp master.
  if ((ActiveThreads % WARPSIZE == 0) ||
      (WarpId < WarpsNeeded - 1)) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else if (ActiveThreads > 1) // Partial warp but contiguous lanes
    // Only SPMD execution mode comes thru this case.
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/ActiveThreads % WARPSIZE,
                              /*LaneId=*/ThreadId % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  if (ActiveThreads > WARPSIZE) {
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded, ThreadId);
  }
#else
  if (ThreadId >= NumTeams)
    return 0;

  // Load from scratchpad and reduce.
  char *scratchpad = GetTeamsReductionScratchpad();
  ldFct(reduce_data, scratchpad, ThreadId, NumTeams, /*Load only*/ 0);
  for (uint32_t i = NumThreads + ThreadId; i < NumTeams; i += NumThreads)
    ldFct(reduce_data, scratchpad, i, NumTeams, /*Load and reduce*/ 1);

  // Reduce across warps to the warp master.
  uint32_t Liveness = __BALLOT_SYNC(0xFFFFFFFF, true);
  if (Liveness == 0xffffffff) // Full warp
    gpu_regular_warp_reduce(reduce_data, shflFct);
  else // Partial warp but contiguous lanes
    gpu_irregular_warp_reduce(reduce_data, shflFct,
                              /*LaneCount=*/__popc(Liveness),
                              /*LaneId=*/ThreadId % WARPSIZE);

  // When we have more than [warpsize] number of threads
  // a block reduction is performed here.
  uint32_t ActiveThreads = NumTeams < NumThreads ? NumTeams : NumThreads;
  if (ActiveThreads > WARPSIZE) {
    uint32_t WarpsNeeded = (ActiveThreads + WARPSIZE - 1) / WARPSIZE;
    // Gather all the reduced values from each warp
    // to the first warp.
    cpyFct(reduce_data, WarpsNeeded);

    uint32_t WarpId = ThreadId / WARPSIZE;
    if (WarpId == 0)
      gpu_irregular_warp_reduce(reduce_data, shflFct, WarpsNeeded, ThreadId);
  }
#endif // __CUDA_ARCH__ >= 700

  return ThreadId == 0;
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait(int32_t global_tid, int32_t num_vars,
                                         size_t reduce_size, void *reduce_data,
                                         kmp_ShuffleReductFctPtr shflFct,
                                         kmp_InterWarpCopyFctPtr cpyFct,
                                         kmp_CopyToScratchpadFctPtr scratchFct,
                                         kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct, /*isSPMDExecutionMode=*/isSPMDMode());
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait_simple_spmd(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    kmp_CopyToScratchpadFctPtr scratchFct, kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct,
                                   /*isSPMDExecutionMode=*/true,
                                   /*isRuntimeUninitialized=*/true);
}

EXTERN
int32_t __kmpc_nvptx_teams_reduce_nowait_simple_generic(
    int32_t global_tid, int32_t num_vars, size_t reduce_size, void *reduce_data,
    kmp_ShuffleReductFctPtr shflFct, kmp_InterWarpCopyFctPtr cpyFct,
    kmp_CopyToScratchpadFctPtr scratchFct, kmp_LoadReduceFctPtr ldFct) {
  return nvptx_teams_reduce_nowait(global_tid, num_vars, reduce_size,
                                   reduce_data, shflFct, cpyFct, scratchFct,
                                   ldFct,
                                   /*isSPMDExecutionMode=*/false,
                                   /*isRuntimeUninitialized=*/true);
}
