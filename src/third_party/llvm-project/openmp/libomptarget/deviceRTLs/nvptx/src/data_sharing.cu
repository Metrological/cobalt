#include "hip/hip_runtime.h"
//===----- data_sharing.cu - NVPTX OpenMP debug utilities -------- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of data sharing environments/
//
//===----------------------------------------------------------------------===//
#include "omptarget-nvptx.h"
#include <stdio.h>

// Number of threads in the CUDA block.
__device__ static unsigned getNumThreads() { return blockDim.x; }
// Thread ID in the CUDA block
__device__ static unsigned getThreadId() { return threadIdx.x; }
// Warp ID in the CUDA block
__device__ static unsigned getWarpId() { return threadIdx.x / WARPSIZE; }
// Lane ID in the CUDA warp.
__device__ static unsigned getLaneId() { return threadIdx.x % WARPSIZE; }

// The CUDA thread ID of the master thread.
__device__ static unsigned getMasterThreadId() {
  unsigned Mask = WARPSIZE - 1;
  return (getNumThreads() - 1) & (~Mask);
}

// Find the active threads in the warp - return a mask whose n-th bit is set if
// the n-th thread in the warp is active.
__device__ static unsigned getActiveThreadsMask() {
  return __BALLOT_SYNC(0xFFFFFFFF, true);
}

// Return true if this is the first active thread in the warp.
__device__ static bool IsWarpMasterActiveThread() {
  unsigned long long Mask = getActiveThreadsMask();
  unsigned long long ShNum = WARPSIZE - (getThreadId() % WARPSIZE);
  unsigned long long Sh = Mask << ShNum;
  // Truncate Sh to the 32 lower bits
  return (unsigned)Sh == 0;
}
// Return true if this is the master thread.
__device__ static bool IsMasterThread() {
  return !isSPMDMode() && getMasterThreadId() == getThreadId();
}

/// Return the provided size aligned to the size of a pointer.
__device__ static size_t AlignVal(size_t Val) {
  const size_t Align = (size_t)sizeof(void *);
  if (Val & (Align - 1)) {
    Val += Align;
    Val &= ~(Align - 1);
  }
  return Val;
}

#define DSFLAG 0
#define DSFLAG_INIT 0
#define DSPRINT(_flag, _str, _args...)                                         \
  {                                                                            \
    if (_flag) {                                                               \
      /*printf("(%d,%d) -> " _str, blockIdx.x, threadIdx.x, _args);*/          \
    }                                                                          \
  }
#define DSPRINT0(_flag, _str)                                                  \
  {                                                                            \
    if (_flag) {                                                               \
      /*printf("(%d,%d) -> " _str, blockIdx.x, threadIdx.x);*/                 \
    }                                                                          \
  }

// Initialize the shared data structures. This is expected to be called for the
// master thread and warp masters. \param RootS: A pointer to the root of the
// data sharing stack. \param InitialDataSize: The initial size of the data in
// the slot.
EXTERN void
__kmpc_initialize_data_sharing_environment(__kmpc_data_sharing_slot *rootS,
                                           size_t InitialDataSize) {

  DSPRINT0(DSFLAG_INIT,
           "Entering __kmpc_initialize_data_sharing_environment\n");

  unsigned WID = getWarpId();
  DSPRINT(DSFLAG_INIT, "Warp ID: %d\n", WID);

  omptarget_nvptx_TeamDescr *teamDescr =
      &omptarget_nvptx_threadPrivateContext->TeamContext();
  __kmpc_data_sharing_slot *RootS = teamDescr->RootS(WID, IsMasterThread());

  DataSharingState.SlotPtr[WID] = RootS;
  DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];

  // We don't need to initialize the frame and active threads.

  DSPRINT(DSFLAG_INIT, "Initial data size: %08x \n", InitialDataSize);
  DSPRINT(DSFLAG_INIT, "Root slot at: %016llx \n", (long long)RootS);
  DSPRINT(DSFLAG_INIT, "Root slot data-end at: %016llx \n",
          (long long)RootS->DataEnd);
  DSPRINT(DSFLAG_INIT, "Root slot next at: %016llx \n", (long long)RootS->Next);
  DSPRINT(DSFLAG_INIT, "Shared slot ptr at: %016llx \n",
          (long long)DataSharingState.SlotPtr[WID]);
  DSPRINT(DSFLAG_INIT, "Shared stack ptr at: %016llx \n",
          (long long)DataSharingState.StackPtr[WID]);

  DSPRINT0(DSFLAG_INIT, "Exiting __kmpc_initialize_data_sharing_environment\n");
}

EXTERN void *__kmpc_data_sharing_environment_begin(
    __kmpc_data_sharing_slot **SavedSharedSlot, void **SavedSharedStack,
    void **SavedSharedFrame, int32_t *SavedActiveThreads,
    size_t SharingDataSize, size_t SharingDefaultDataSize,
    int16_t IsOMPRuntimeInitialized) {

  DSPRINT0(DSFLAG, "Entering __kmpc_data_sharing_environment_begin\n");

  // If the runtime has been elided, used __shared__ memory for master-worker
  // data sharing.
  if (!IsOMPRuntimeInitialized)
    return (void *)&DataSharingState;

  DSPRINT(DSFLAG, "Data Size %016llx\n", SharingDataSize);
  DSPRINT(DSFLAG, "Default Data Size %016llx\n", SharingDefaultDataSize);

  unsigned WID = getWarpId();
  unsigned CurActiveThreads = getActiveThreadsMask();

  __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
  void *&StackP = DataSharingState.StackPtr[WID];
  void *&FrameP = DataSharingState.FramePtr[WID];
  int32_t &ActiveT = DataSharingState.ActiveThreads[WID];

  DSPRINT0(DSFLAG, "Save current slot/stack values.\n");
  // Save the current values.
  *SavedSharedSlot = SlotP;
  *SavedSharedStack = StackP;
  *SavedSharedFrame = FrameP;
  *SavedActiveThreads = ActiveT;

  DSPRINT(DSFLAG, "Warp ID: %d\n", WID);
  DSPRINT(DSFLAG, "Saved slot ptr at: %016llx \n", (long long)SlotP);
  DSPRINT(DSFLAG, "Saved stack ptr at: %016llx \n", (long long)StackP);
  DSPRINT(DSFLAG, "Saved frame ptr at: %016llx \n", (long long)FrameP);
  DSPRINT(DSFLAG, "Active threads: %08x \n", ActiveT);

  // Only the warp active master needs to grow the stack.
  if (IsWarpMasterActiveThread()) {
    // Save the current active threads.
    ActiveT = CurActiveThreads;

    // Make sure we use aligned sizes to avoid rematerialization of data.
    SharingDataSize = AlignVal(SharingDataSize);
    // FIXME: The default data size can be assumed to be aligned?
    SharingDefaultDataSize = AlignVal(SharingDefaultDataSize);

    // Check if we have room for the data in the current slot.
    const uintptr_t CurrentStartAddress = (uintptr_t)StackP;
    const uintptr_t CurrentEndAddress = (uintptr_t)SlotP->DataEnd;
    const uintptr_t RequiredEndAddress =
        CurrentStartAddress + (uintptr_t)SharingDataSize;

    DSPRINT(DSFLAG, "Data Size %016llx\n", SharingDataSize);
    DSPRINT(DSFLAG, "Default Data Size %016llx\n", SharingDefaultDataSize);
    DSPRINT(DSFLAG, "Current Start Address %016llx\n", CurrentStartAddress);
    DSPRINT(DSFLAG, "Current End Address %016llx\n", CurrentEndAddress);
    DSPRINT(DSFLAG, "Required End Address %016llx\n", RequiredEndAddress);
    DSPRINT(DSFLAG, "Active Threads %08x\n", ActiveT);

    // If we require a new slot, allocate it and initialize it (or attempt to
    // reuse one). Also, set the shared stack and slot pointers to the new
    // place. If we do not need to grow the stack, just adapt the stack and
    // frame pointers.
    if (CurrentEndAddress < RequiredEndAddress) {
      size_t NewSize = (SharingDataSize > SharingDefaultDataSize)
                           ? SharingDataSize
                           : SharingDefaultDataSize;
      __kmpc_data_sharing_slot *NewSlot = 0;

      // Attempt to reuse an existing slot.
      if (__kmpc_data_sharing_slot *ExistingSlot = SlotP->Next) {
        uintptr_t ExistingSlotSize = (uintptr_t)ExistingSlot->DataEnd -
                                     (uintptr_t)(&ExistingSlot->Data[0]);
        if (ExistingSlotSize >= NewSize) {
          DSPRINT(DSFLAG, "Reusing stack slot %016llx\n",
                  (long long)ExistingSlot);
          NewSlot = ExistingSlot;
        } else {
          DSPRINT(DSFLAG, "Cleaning up -failed reuse - %016llx\n",
                  (long long)SlotP->Next);
          free(ExistingSlot);
        }
      }

      if (!NewSlot) {
        NewSlot = (__kmpc_data_sharing_slot *)malloc(
            sizeof(__kmpc_data_sharing_slot) + NewSize);
        DSPRINT(DSFLAG, "New slot allocated %016llx (data size=%016llx)\n",
                (long long)NewSlot, NewSize);
      }

      NewSlot->Next = 0;
      NewSlot->DataEnd = &NewSlot->Data[NewSize];

      SlotP->Next = NewSlot;
      SlotP = NewSlot;
      StackP = &NewSlot->Data[SharingDataSize];
      FrameP = &NewSlot->Data[0];
    } else {

      // Clean up any old slot that we may still have. The slot producers, do
      // not eliminate them because that may be used to return data.
      if (SlotP->Next) {
        DSPRINT(DSFLAG, "Cleaning up - old not required - %016llx\n",
                (long long)SlotP->Next);
        free(SlotP->Next);
        SlotP->Next = 0;
      }

      FrameP = StackP;
      StackP = (void *)RequiredEndAddress;
    }
  }

  // FIXME: Need to see the impact of doing it here.
  __threadfence_block();

  DSPRINT0(DSFLAG, "Exiting __kmpc_data_sharing_environment_begin\n");

  // All the threads in this warp get the frame they should work with.
  return FrameP;
}

EXTERN void __kmpc_data_sharing_environment_end(
    __kmpc_data_sharing_slot **SavedSharedSlot, void **SavedSharedStack,
    void **SavedSharedFrame, int32_t *SavedActiveThreads,
    int32_t IsEntryPoint) {

  DSPRINT0(DSFLAG, "Entering __kmpc_data_sharing_environment_end\n");

  unsigned WID = getWarpId();

  if (IsEntryPoint) {
    if (IsWarpMasterActiveThread()) {
      DSPRINT0(DSFLAG, "Doing clean up\n");

      // The master thread cleans the saved slot, because this is an environment
      // only for the master.
      __kmpc_data_sharing_slot *S =
          IsMasterThread() ? *SavedSharedSlot : DataSharingState.SlotPtr[WID];

      if (S->Next) {
        free(S->Next);
        S->Next = 0;
      }
    }

    DSPRINT0(DSFLAG, "Exiting Exiting __kmpc_data_sharing_environment_end\n");
    return;
  }

  int32_t CurActive = getActiveThreadsMask();

  // Only the warp master can restore the stack and frame information, and only
  // if there are no other threads left behind in this environment (i.e. the
  // warp diverged and returns in different places). This only works if we
  // assume that threads will converge right after the call site that started
  // the environment.
  if (IsWarpMasterActiveThread()) {
    int32_t &ActiveT = DataSharingState.ActiveThreads[WID];

    DSPRINT0(DSFLAG, "Before restoring the stack\n");
    // Zero the bits in the mask. If it is still different from zero, then we
    // have other threads that will return after the current ones.
    ActiveT &= ~CurActive;

    DSPRINT(DSFLAG, "Active threads: %08x; New mask: %08x\n", CurActive,
            ActiveT);

    if (!ActiveT) {
      // No other active threads? Great, lets restore the stack.

      __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
      void *&StackP = DataSharingState.StackPtr[WID];
      void *&FrameP = DataSharingState.FramePtr[WID];

      SlotP = *SavedSharedSlot;
      StackP = *SavedSharedStack;
      FrameP = *SavedSharedFrame;
      ActiveT = *SavedActiveThreads;

      DSPRINT(DSFLAG, "Restored slot ptr at: %016llx \n", (long long)SlotP);
      DSPRINT(DSFLAG, "Restored stack ptr at: %016llx \n", (long long)StackP);
      DSPRINT(DSFLAG, "Restored frame ptr at: %016llx \n", (long long)FrameP);
      DSPRINT(DSFLAG, "Active threads: %08x \n", ActiveT);
    }
  }

  // FIXME: Need to see the impact of doing it here.
  __threadfence_block();

  DSPRINT0(DSFLAG, "Exiting __kmpc_data_sharing_environment_end\n");
  return;
}

EXTERN void *
__kmpc_get_data_sharing_environment_frame(int32_t SourceThreadID,
                                          int16_t IsOMPRuntimeInitialized) {
  DSPRINT0(DSFLAG, "Entering __kmpc_get_data_sharing_environment_frame\n");

  // If the runtime has been elided, use __shared__ memory for master-worker
  // data sharing.  We're reusing the statically allocated data structure
  // that is used for standard data sharing.
  if (!IsOMPRuntimeInitialized)
    return (void *)&DataSharingState;

  // Get the frame used by the requested thread.

  unsigned SourceWID = SourceThreadID / WARPSIZE;

  DSPRINT(DSFLAG, "Source  warp: %d\n", SourceWID);

  void *P = DataSharingState.FramePtr[SourceWID];
  DSPRINT0(DSFLAG, "Exiting __kmpc_get_data_sharing_environment_frame\n");
  return P;
}

////////////////////////////////////////////////////////////////////////////////
// Runtime functions for trunk data sharing scheme.
////////////////////////////////////////////////////////////////////////////////

INLINE void data_sharing_init_stack_common() {
  omptarget_nvptx_TeamDescr *teamDescr =
      &omptarget_nvptx_threadPrivateContext->TeamContext();

  for (int WID = 0; WID < WARPSIZE; WID++) {
    __kmpc_data_sharing_slot *RootS = teamDescr->GetPreallocatedSlotAddr(WID);
    DataSharingState.SlotPtr[WID] = RootS;
    DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];
  }
}

// Initialize data sharing data structure. This function needs to be called
// once at the beginning of a data sharing context (coincides with the kernel
// initialization). This function is called only by the MASTER thread of each
// team in non-SPMD mode.
EXTERN void __kmpc_data_sharing_init_stack() {
  // This function initializes the stack pointer with the pointer to the
  // statically allocated shared memory slots. The size of a shared memory
  // slot is pre-determined to be 256 bytes.
  data_sharing_init_stack_common();
  omptarget_nvptx_globalArgs.Init();
}

// Initialize data sharing data structure. This function needs to be called
// once at the beginning of a data sharing context (coincides with the kernel
// initialization). This function is called in SPMD mode only.
EXTERN void __kmpc_data_sharing_init_stack_spmd() {
  // This function initializes the stack pointer with the pointer to the
  // statically allocated shared memory slots. The size of a shared memory
  // slot is pre-determined to be 256 bytes.
  if (threadIdx.x == 0)
    data_sharing_init_stack_common();

  __threadfence_block();
}

// Called at the time of the kernel initialization. This is used to initilize
// the list of references to shared variables and to pre-allocate global storage
// for holding the globalized variables.
//
// By default the globalized variables are stored in global memory. If the
// UseSharedMemory is set to true, the runtime will attempt to use shared memory
// as long as the size requested fits the pre-allocated size.
EXTERN void* __kmpc_data_sharing_push_stack(size_t DataSize,
    int16_t UseSharedMemory) {
  // Frame pointer must be visible to all workers in the same warp.
  unsigned WID = getWarpId();
  void *&FrameP = DataSharingState.FramePtr[WID];

  // Only warp active master threads manage the stack.
  if (IsWarpMasterActiveThread()) {
    // SlotP will point to either the shared memory slot or an existing
    // global memory slot.
    __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
    void *&StackP = DataSharingState.StackPtr[WID];

    // Compute the total memory footprint of the requested data.
    // The master thread requires a stack only for itself. A worker
    // thread (which at this point is a warp master) will require
    // space for the variables of each thread in the warp,
    // i.e. one DataSize chunk per warp lane.
    // TODO: change WARPSIZE to the number of active threads in the warp.
    size_t PushSize = IsMasterThread() ? DataSize : WARPSIZE * DataSize;

    // Check if we have room for the data in the current slot.
    const uintptr_t StartAddress = (uintptr_t)StackP;
    const uintptr_t EndAddress = (uintptr_t)SlotP->DataEnd;
    const uintptr_t RequestedEndAddress = StartAddress + (uintptr_t)PushSize;

    // If we requested more data than there is room for in the rest
    // of the slot then we need to either re-use the next slot, if one exists,
    // or create a new slot.
    if (EndAddress < RequestedEndAddress) {
      __kmpc_data_sharing_slot *NewSlot = 0;
      size_t NewSize = PushSize;

      // Allocate at least the default size for each type of slot.
      // Master is a special case and even though there is only one thread,
      // it can share more things with the workers. For uniformity, it uses
      // the full size of a worker warp slot.
      size_t DefaultSlotSize = DS_Worker_Warp_Slot_Size;
      if (DefaultSlotSize > NewSize)
        NewSize = DefaultSlotSize;
      NewSlot = (__kmpc_data_sharing_slot *) SafeMalloc(
          sizeof(__kmpc_data_sharing_slot) + NewSize,
          "Global memory slot allocation.");

      NewSlot->Next = 0;
      NewSlot->Prev = SlotP;
      NewSlot->PrevSlotStackPtr = StackP;
      NewSlot->DataEnd = &NewSlot->Data[0] + NewSize;

      // Make previous slot point to the newly allocated slot.
      SlotP->Next = NewSlot;
      // The current slot becomes the new slot.
      SlotP = NewSlot;
      // The stack pointer always points to the next free stack frame.
      StackP = &NewSlot->Data[0] + PushSize;
      // The frame pointer always points to the beginning of the frame.
      FrameP = &NewSlot->Data[0];
    } else {
      // Add the data chunk to the current slot. The frame pointer is set to
      // point to the start of the new frame held in StackP.
      FrameP = StackP;
      // Reset stack pointer to the requested address.
      StackP = (void *)RequestedEndAddress;
    }
  }

  __threadfence_block();

  // Compute the start address of the frame of each thread in the warp.
  uintptr_t FrameStartAddress = (uintptr_t)FrameP;
  FrameStartAddress += (uintptr_t) (getLaneId() * DataSize);
  return (void *)FrameStartAddress;
}

// Pop the stack and free any memory which can be reclaimed.
//
// When the pop operation removes the last global memory slot,
// reclaim all outstanding global memory slots since it is
// likely we have reached the end of the kernel.
EXTERN void __kmpc_data_sharing_pop_stack(void *FrameStart) {
  if (IsWarpMasterActiveThread()) {
    unsigned WID = getWarpId();

    // Current slot
    __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];

    // Pointer to next available stack.
    void *&StackP = DataSharingState.StackPtr[WID];

    // If the current slot is empty, we need to free the slot after the
    // pop.
    bool SlotEmpty = (StackP == &SlotP->Data[0]);

    // Pop the frame.
    StackP = FrameStart;

    if (SlotEmpty && SlotP->Prev) {
      // Before removing the slot we need to reset StackP.
      StackP = SlotP->PrevSlotStackPtr;

      // Remove the slot.
      SlotP = SlotP->Prev;
      SafeFree(SlotP->Next, "Free slot.");
      SlotP->Next = 0;
    }
  }

  __threadfence_block();
}

// Begin a data sharing context. Maintain a list of references to shared
// variables. This list of references to shared variables will be passed
// to one or more threads.
// In L0 data sharing this is called by master thread.
// In L1 data sharing this is called by active warp master thread.
EXTERN void __kmpc_begin_sharing_variables(void ***GlobalArgs, size_t nArgs) {
  omptarget_nvptx_globalArgs.EnsureSize(nArgs);
  *GlobalArgs = omptarget_nvptx_globalArgs.GetArgs();
}

// End a data sharing context. There is no need to have a list of refs
// to shared variables because the context in which those variables were
// shared has now ended. This should clean-up the list of references only
// without affecting the actual global storage of the variables.
// In L0 data sharing this is called by master thread.
// In L1 data sharing this is called by active warp master thread.
EXTERN void __kmpc_end_sharing_variables() {
  omptarget_nvptx_globalArgs.DeInit();
}

// This function will return a list of references to global variables. This
// is how the workers will get a reference to the globalized variable. The
// members of this list will be passed to the outlined parallel function
// preserving the order.
// Called by all workers.
EXTERN void __kmpc_get_shared_variables(void ***GlobalArgs) {
  *GlobalArgs = omptarget_nvptx_globalArgs.GetArgs();
}
