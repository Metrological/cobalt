#include "hip/hip_runtime.h"
//===------------ sync.h - NVPTX OpenMP synchronizations --------- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// Include all synchronization.
//
//===----------------------------------------------------------------------===//

#include "omptarget-nvptx.h"

////////////////////////////////////////////////////////////////////////////////
// KMP Ordered calls
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_ordered(kmp_Indent *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_ordered\n");
}

EXTERN void __kmpc_end_ordered(kmp_Indent *loc, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_end_ordered\n");
}

////////////////////////////////////////////////////////////////////////////////
// KMP Barriers
////////////////////////////////////////////////////////////////////////////////

// a team is a block: we can use CUDA native synchronization mechanism
// FIXME: what if not all threads (warps) participate to the barrier?
// We may need to implement it differently

EXTERN int32_t __kmpc_cancel_barrier(kmp_Indent *loc_ref, int32_t tid) {
  PRINT0(LD_IO, "call kmpc_cancel_barrier\n");
  __kmpc_barrier(loc_ref, tid);
  PRINT0(LD_SYNC, "completed kmpc_cancel_barrier\n");
  return 0;
}

EXTERN void __kmpc_barrier(kmp_Indent *loc_ref, int32_t tid) {
  if (isRuntimeUninitialized()) {
    if (isSPMDMode())
      __kmpc_barrier_simple_spmd(loc_ref, tid);
    else
      __kmpc_barrier_simple_generic(loc_ref, tid);
  } else {
    tid = GetLogicalThreadIdInBlock();
    omptarget_nvptx_TaskDescr *currTaskDescr =
        omptarget_nvptx_threadPrivateContext->GetTopLevelTaskDescr(tid);
    int numberOfActiveOMPThreads = GetNumberOfOmpThreads(
        tid, isSPMDMode(), /*isRuntimeUninitialized=*/false);
    if (numberOfActiveOMPThreads > 1) {
      if (isSPMDMode()) {
        __kmpc_barrier_simple_spmd(loc_ref, tid);
      } else {
        // The #threads parameter must be rounded up to the WARPSIZE.
        int threads =
            WARPSIZE * ((numberOfActiveOMPThreads + WARPSIZE - 1) / WARPSIZE);

        PRINT(LD_SYNC,
              "call kmpc_barrier with %d omp threads, sync parameter %d\n",
              numberOfActiveOMPThreads, threads);
        // Barrier #1 is for synchronization among active threads.
        named_sync(L1_BARRIER, threads);
      }
    } // numberOfActiveOMPThreads > 1
    PRINT0(LD_SYNC, "completed kmpc_barrier\n");
  }
}

// Emit a simple barrier call in SPMD mode.  Assumes the caller is in an L0
// parallel region and that all worker threads participate.
EXTERN void __kmpc_barrier_simple_spmd(kmp_Indent *loc_ref, int32_t tid) {
  PRINT0(LD_SYNC, "call kmpc_barrier_simple_spmd\n");
  __syncthreads();
  PRINT0(LD_SYNC, "completed kmpc_barrier_simple_spmd\n");
}

// Emit a simple barrier call in Generic mode.  Assumes the caller is in an L0
// parallel region and that all worker threads participate.
EXTERN void __kmpc_barrier_simple_generic(kmp_Indent *loc_ref, int32_t tid) {
  int numberOfActiveOMPThreads = GetNumberOfThreadsInBlock() - WARPSIZE;
  // The #threads parameter must be rounded up to the WARPSIZE.
  int threads =
      WARPSIZE * ((numberOfActiveOMPThreads + WARPSIZE - 1) / WARPSIZE);

  PRINT(LD_SYNC,
        "call kmpc_barrier_simple_generic with %d omp threads, sync parameter "
        "%d\n",
        numberOfActiveOMPThreads, threads);
  // Barrier #1 is for synchronization among active threads.
  named_sync(L1_BARRIER, threads);
  PRINT0(LD_SYNC, "completed kmpc_barrier_simple_generic\n");
}

////////////////////////////////////////////////////////////////////////////////
// KMP MASTER
////////////////////////////////////////////////////////////////////////////////

INLINE int32_t IsMaster() {
  // only the team master updates the state
  int tid = GetLogicalThreadIdInBlock();
  int ompThreadId = GetOmpThreadId(tid, isSPMDMode(), isRuntimeUninitialized());
  return IsTeamMaster(ompThreadId);
}

EXTERN int32_t __kmpc_master(kmp_Indent *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_master\n");
  return IsMaster();
}

EXTERN void __kmpc_end_master(kmp_Indent *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_end_master\n");
  ASSERT0(LT_FUSSY, IsMaster(), "expected only master here");
}

////////////////////////////////////////////////////////////////////////////////
// KMP SINGLE
////////////////////////////////////////////////////////////////////////////////

EXTERN int32_t __kmpc_single(kmp_Indent *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_single\n");
  // decide to implement single with master; master get the single
  return IsMaster();
}

EXTERN void __kmpc_end_single(kmp_Indent *loc, int32_t global_tid) {
  PRINT0(LD_IO, "call kmpc_end_single\n");
  // decide to implement single with master: master get the single
  ASSERT0(LT_FUSSY, IsMaster(), "expected only master here");
  // sync barrier is explicitely called... so that is not a problem
}

////////////////////////////////////////////////////////////////////////////////
// Flush
////////////////////////////////////////////////////////////////////////////////

EXTERN void __kmpc_flush(kmp_Indent *loc) {
  PRINT0(LD_IO, "call kmpc_flush\n");
  __threadfence_block();
}

////////////////////////////////////////////////////////////////////////////////
// Vote
////////////////////////////////////////////////////////////////////////////////

EXTERN int32_t __kmpc_warp_active_thread_mask() {
  PRINT0(LD_IO, "call __kmpc_warp_active_thread_mask\n");
  return __ACTIVEMASK();
}
